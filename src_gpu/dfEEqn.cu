#include "dfEEqn.H"

double* dfEEqn::getFieldPointer(const char* fieldAlias, location loc, position pos) {
    char mergedName[256];
    if (pos == position::internal) {
        sprintf(mergedName, "%s_%s", (loc == location::cpu) ? "h" : "d", fieldAlias);
    } else if (pos == position::boundary) {
        sprintf(mergedName, "%s_boundary_%s", (loc == location::cpu) ? "h" : "d", fieldAlias);
    }

    double *pointer = nullptr;
    if (fieldPointerMap.find(std::string(mergedName)) != fieldPointerMap.end()) {
        pointer = fieldPointerMap[std::string(mergedName)];
    }
    if (pointer == nullptr) {
        fprintf(stderr, "Warning! getFieldPointer of %s returns nullptr!\n", mergedName);
    }
    //fprintf(stderr, "fieldAlias: %s, mergedName: %s, pointer: %p\n", fieldAlias, mergedName, pointer);

    return pointer;
}

void dfEEqn::setConstantValues(const std::string &mode_string, const std::string &setting_path) {
    this->stream = dataBase_.stream;
    this->mode_string = mode_string;
    this->setting_path = setting_path;
    ESolver = new AmgXSolver(mode_string, setting_path, dataBase_.localRank);
}

void dfEEqn::setConstantFields(const std::vector<int> patch_type_he, const std::vector<int> patch_type_k) {
    this->patch_type_he = patch_type_he;
    this->patch_type_k = patch_type_k;
    // calculate num_gradientEnergy_boundary_surfaces
    for (int i = 0; i < dataBase_.num_patches; i++) {
        if (patch_type_he[i] == boundaryConditions::gradientEnergy) {
            num_gradientEnergy_boundary_surfaces += dataBase_.patch_size[i];
        }
    }
}

void dfEEqn::createNonConstantFieldsInternal() {
#ifndef STREAM_ALLOCATOR
    // thermophysical fields
    checkCudaErrors(hipMalloc((void**)&d_dpdt, dataBase_.cell_value_bytes));
    // boundary coeffs
    checkCudaErrors(hipMalloc((void**)&d_value_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_value_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_gradient_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_gradient_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
#endif
    // computed on CPU, used on GPU, need memcpyh2d
    checkCudaErrors(hipHostMalloc((void**)&h_dpdt, dataBase_.cell_value_bytes));

    // getter for h_dpdt
    fieldPointerMap["h_dpdt"] = h_dpdt;
}

void dfEEqn::createNonConstantFieldsBoundary() {
#ifndef STREAM_ALLOCATOR
    checkCudaErrors(hipMalloc((void**)&d_boundary_heGradient, sizeof(double) * num_gradientEnergy_boundary_surfaces));
#endif
    // computed on CPU, used on GPU, need memcpyh2d
    checkCudaErrors(hipHostMalloc((void**)&h_boundary_heGradient, sizeof(double) * num_gradientEnergy_boundary_surfaces));

    // getter for h_boundary_heGradient
    fieldPointerMap["h_boundary_heGradient"] = h_boundary_heGradient;
}


void dfEEqn::createNonConstantLduAndCsrFields() {
    checkCudaErrors(hipMalloc((void**)&d_ldu, dataBase_.csr_value_bytes));
    d_lower = d_ldu;
    d_diag = d_ldu + dataBase_.num_surfaces;
    d_upper = d_ldu + dataBase_.num_cells + dataBase_.num_surfaces;
    d_extern = d_ldu + dataBase_.num_cells + 2 * dataBase_.num_surfaces;
#ifndef STREAM_ALLOCATOR
    checkCudaErrors(hipMalloc((void**)&d_source, dataBase_.cell_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_internal_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_boundary_coeffs, dataBase_.boundary_surface_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_A, dataBase_.csr_value_bytes));
    checkCudaErrors(hipMalloc((void**)&d_b, dataBase_.cell_value_bytes));
#endif
}

void dfEEqn::initNonConstantFields(const double *he, const double *boundary_he)
{
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_he, he, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_he, boundary_he, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice));
}

void dfEEqn::cleanCudaResources() {
#ifdef USE_GRAPH
    if (pre_graph_created) {
        checkCudaErrors(hipGraphExecDestroy(graph_instance_pre));
        checkCudaErrors(hipGraphDestroy(graph_pre));
    }
    if (post_graph_created) {
        checkCudaErrors(hipGraphExecDestroy(graph_instance_post));
        checkCudaErrors(hipGraphDestroy(graph_post));
    }
#endif
}

void dfEEqn::preProcess(const double *h_he, const double *h_k, const double *h_k_old, const double *h_dpdt, const double *h_boundary_k, const double *h_boundary_heGradient)
{
}

void dfEEqn::process() {
    TICK_INIT_EVENT;
    TICK_START_EVENT;
#ifdef USE_GRAPH
    if(!pre_graph_created) {
        DEBUG_TRACE;
        checkCudaErrors(hipStreamBeginCapture(dataBase_.stream, hipStreamCaptureModeGlobal));
#endif

#ifdef STREAM_ALLOCATOR
    // thermophysical fields
    checkCudaErrors(hipMallocAsync((void**)&d_dpdt, dataBase_.cell_value_bytes, dataBase_.stream));
    // fiv weight fields
    //checkCudaErrors(hipMallocAsync((void**)&d_phi_special_weight, dataBase_.cell_value_bytes, dataBase_.stream));
    // boundary coeffs
    checkCudaErrors(hipMallocAsync((void**)&d_value_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_value_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_gradient_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_gradient_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
 
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_heGradient, sizeof(double) * num_gradientEnergy_boundary_surfaces, dataBase_.stream));

    checkCudaErrors(hipMallocAsync((void**)&d_source, dataBase_.cell_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_internal_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_boundary_coeffs, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_A, dataBase_.csr_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMallocAsync((void**)&d_b, dataBase_.cell_value_bytes, dataBase_.stream));
#endif
    // checkCudaErrors(hipMemcpyAsync(dataBase_.d_k, dataBase_.h_k, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    // checkCudaErrors(hipMemcpyAsync(dataBase_.d_k_old, dataBase_.h_k_old, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    // checkCudaErrors(hipMemcpyAsync(d_dpdt, h_dpdt, dataBase_.cell_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));
    // checkCudaErrors(hipMemcpyAsync(dataBase_.d_boundary_k, dataBase_.h_boundary_k, dataBase_.boundary_surface_value_bytes, hipMemcpyHostToDevice, dataBase_.stream));

    checkCudaErrors(hipMemsetAsync(d_ldu, 0, dataBase_.csr_value_bytes, dataBase_.stream)); // d_ldu contains d_lower, d_diag, and d_upper
    checkCudaErrors(hipMemsetAsync(d_source, 0, dataBase_.cell_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_internal_coeffs, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_boundary_coeffs, 0, dataBase_.boundary_surface_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_A, 0, dataBase_.csr_value_bytes, dataBase_.stream));
    checkCudaErrors(hipMemsetAsync(d_b, 0, dataBase_.cell_value_bytes, dataBase_.stream));

    eeqn_calculate_energy_gradient(thermo_, dataBase_.num_cells, dataBase_.num_species, dataBase_.num_boundary_surfaces, 
            dataBase_.d_boundary_face_cell, dataBase_.d_T, dataBase_.d_p, dataBase_.d_y,
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_he.data(),
            dataBase_.d_boundary_delta_coeffs, dataBase_.d_boundary_p, dataBase_.d_boundary_y,
            d_boundary_heGradient);
    correct_boundary_conditions_scalar(dataBase_.stream, dataBase_.nccl_comm, dataBase_.neighbProcNo.data(),
            dataBase_.num_boundary_surfaces, dataBase_.num_patches, dataBase_.patch_size.data(),
            patch_type_he.data(), dataBase_.d_boundary_delta_coeffs, dataBase_.d_boundary_face_cell,
            dataBase_.d_he, dataBase_.d_boundary_he, dataBase_.cyclicNeighbor.data(), 
            dataBase_.patchSizeOffset.data(), dataBase_.d_boundary_weight,
            dataBase_.d_boundary_T, dataBase_.d_boundary_y, d_boundary_heGradient, &thermo_);
    update_boundary_coeffs_scalar(dataBase_.stream,
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_he.data(),
            dataBase_.d_boundary_delta_coeffs, dataBase_.d_boundary_he, dataBase_.d_boundary_weight,
            d_value_internal_coeffs, d_value_boundary_coeffs,
            d_gradient_internal_coeffs, d_gradient_boundary_coeffs, d_boundary_heGradient);
    fvm_ddt_vol_scalar_vol_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.rdelta_t, dataBase_.d_rho, dataBase_.d_rho_old, 
            dataBase_.d_he, dataBase_.d_volume, d_diag, d_source);
    // NOTE: fvm_div_scalar use d_phi_weight, which is computed in YEqn_GPU by compute_upwind_weight()
    // thus we need open YEqn_GPU before UEqn_GPU
    fvm_div_scalar(dataBase_.stream, dataBase_.num_surfaces, dataBase_.d_owner, dataBase_.d_neighbor,
            dataBase_.d_phi, dataBase_.d_phi_weight,
            d_lower, d_upper, d_diag, // end for internal
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_he.data(),
            dataBase_.d_boundary_phi,
            d_value_internal_coeffs, d_value_boundary_coeffs,
            d_internal_coeffs, d_boundary_coeffs, 1.);
    fvc_ddt_vol_scalar_vol_scalar(dataBase_.stream, dataBase_.num_cells,
            dataBase_.rdelta_t, dataBase_.d_rho, dataBase_.d_rho_old, dataBase_.d_k,
            dataBase_.d_k_old, dataBase_.d_volume, d_source, -1.);
    fvc_div_surface_scalar_vol_scalar(dataBase_.stream, dataBase_.num_surfaces, dataBase_.d_owner, dataBase_.d_neighbor, dataBase_.d_weight, 
            dataBase_.d_k, dataBase_.d_phi, d_source, // end for internal
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_k.data(), 
            dataBase_.d_boundary_face_cell, dataBase_.d_boundary_k, dataBase_.d_boundary_phi, -1);
    fvm_laplacian_scalar(dataBase_.stream, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces, dataBase_.d_owner, dataBase_.d_neighbor,
            dataBase_.d_weight, dataBase_.d_mag_sf, dataBase_.d_delta_coeffs, dataBase_.d_thermo_alpha, 
            d_lower, d_upper, d_diag, // end for internal
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_he.data(), dataBase_.d_boundary_mag_sf, dataBase_.d_boundary_thermo_alpha,
            d_gradient_internal_coeffs, d_gradient_boundary_coeffs, d_internal_coeffs, d_boundary_coeffs, -1);
    fvc_div_cell_vector(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces, 
            dataBase_.d_owner, dataBase_.d_neighbor, 
            dataBase_.d_weight, dataBase_.d_sf, dataBase_.d_hDiff_corr_flux, d_source,
            dataBase_.num_patches, dataBase_.patch_size.data(), patch_type_he.data(), dataBase_.d_boundary_face_cell,
            dataBase_.d_boundary_weight, dataBase_.d_boundary_hDiff_corr_flux, dataBase_.d_boundary_sf, dataBase_.d_volume);
    fvc_to_source_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.d_volume, dataBase_.d_dpdt, d_source);
    fvc_to_source_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.d_volume, dataBase_.d_diff_alphaD, d_source, -1);
#ifndef DEBUG_CHECK_LDU
    ldu_to_csr_scalar(dataBase_.stream, dataBase_.num_cells, dataBase_.num_surfaces, dataBase_.num_boundary_surfaces,
            dataBase_.num_Nz, dataBase_.d_boundary_face_cell, dataBase_.d_ldu_to_csr_index, dataBase_.num_patches,
            dataBase_.patch_size.data(), patch_type_he.data(), d_ldu, d_source, d_internal_coeffs, d_boundary_coeffs, d_A);
#endif
#ifdef USE_GRAPH
        checkCudaErrors(hipStreamEndCapture(dataBase_.stream, &graph_pre));
        checkCudaErrors(hipGraphInstantiate(&graph_instance_pre, graph_pre, NULL, NULL, 0));
        pre_graph_created = true;
    }
    DEBUG_TRACE;
    checkCudaErrors(hipGraphLaunch(graph_instance_pre, dataBase_.stream));
#endif
    TICK_END_EVENT(EEqn assembly);

    TICK_START_EVENT;
#ifndef DEBUG_CHECK_LDU
    solve();
#endif
    TICK_END_EVENT(EEqn solve);

#ifdef USE_GRAPH
    if(!post_graph_created) {
        checkCudaErrors(hipStreamBeginCapture(dataBase_.stream, hipStreamCaptureModeGlobal));
#endif

        TICK_START_EVENT;
        correct_boundary_conditions_scalar(dataBase_.stream, dataBase_.nccl_comm, dataBase_.neighbProcNo.data(),
                dataBase_.num_boundary_surfaces, dataBase_.num_patches, dataBase_.patch_size.data(),
                patch_type_he.data(), dataBase_.d_boundary_delta_coeffs, dataBase_.d_boundary_face_cell,
                dataBase_.d_he, dataBase_.d_boundary_he, dataBase_.cyclicNeighbor.data(), 
                dataBase_.patchSizeOffset.data(), dataBase_.d_boundary_weight,
                dataBase_.d_boundary_T, dataBase_.d_boundary_y, d_boundary_heGradient, &thermo_);
        TICK_END_EVENT(EEqn post process correctBC);

        TICK_START_EVENT;
        // copy he to host
        // checkCudaErrors(hipMemcpyAsync(dataBase_.h_he, dataBase_.d_he, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost, dataBase_.stream));
        // checkCudaErrors(hipMemcpyAsync(dataBase_.h_boundary_he, dataBase_.d_boundary_he, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost, dataBase_.stream));
        TICK_END_EVENT(EEqn post process copy back);

        TICK_START_EVENT;
#ifdef STREAM_ALLOCATOR
        // thermophysical fields
        checkCudaErrors(hipFreeAsync(d_dpdt, dataBase_.stream));
        // fiv weight fieldsFree
        //checkCudaErrors(hipFreeAsync(d_phi_special_weight, dataBase_.stream));
        // boundary coeffs
        checkCudaErrors(hipFreeAsync(d_value_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_value_boundary_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_gradient_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_gradient_boundary_coeffs, dataBase_.stream));
     
        checkCudaErrors(hipFreeAsync(d_boundary_heGradient, dataBase_.stream));
    
        checkCudaErrors(hipFreeAsync(d_source, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_internal_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_boundary_coeffs, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_A, dataBase_.stream));
        checkCudaErrors(hipFreeAsync(d_b, dataBase_.stream));
#endif
        TICK_END_EVENT(EEqn post process free);
#ifdef USE_GRAPH
        checkCudaErrors(hipStreamEndCapture(dataBase_.stream, &graph_post));
        checkCudaErrors(hipGraphInstantiate(&graph_instance_post, graph_post, NULL, NULL, 0));
        post_graph_created = true;
    }
    checkCudaErrors(hipGraphLaunch(graph_instance_post, dataBase_.stream));
#endif
    sync();
}

void dfEEqn::eeqn_calculate_energy_gradient(dfThermo& GPUThermo, int num_cells, int num_species, 
        int num_boundary_surfaces, const int *face2Cells, double *T, double *p, double *y,
        int num_patches, const int *patch_size, const int *patch_type,
        const double *boundary_delta_coeffs, const double *boundary_p, const double* boundary_y, 
        double *boundary_thermo_gradient)
{
    int bou_offset = 0, gradient_offset = 0;
    for (int i = 0; i < num_patches; i++) {
        if (patch_size[i] == 0) continue;
        if (patch_type[i] == boundaryConditions::gradientEnergy) {
            GPUThermo.calculateEnergyGradient(patch_size[i], num_cells, num_species, num_boundary_surfaces, bou_offset, gradient_offset,
                    face2Cells, T, p, y, boundary_delta_coeffs, boundary_p, boundary_y, boundary_thermo_gradient);
            bou_offset += patch_size[i];
            gradient_offset += patch_size[i];
        } else if (patch_type[i] == boundaryConditions::processor
                    || patch_type[i] == boundaryConditions::processorCyclic) {
            bou_offset += 2 * patch_size[i];
        } else {
            bou_offset += patch_size[i];
        }
    }
}

// #if defined DEBUG_
void dfEEqn::compareResult(const double *lower, const double *upper, const double *diag, 
        const double *source, const double *internal_coeffs, const double *boundary_coeffs, bool printFlag)
{
    DEBUG_TRACE;
    std::vector<double> h_lower;
    h_lower.resize(dataBase_.num_surfaces);
    checkCudaErrors(hipMemcpy(h_lower.data(), d_lower, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_lower\n");
    checkVectorEqual(dataBase_.num_surfaces, lower, h_lower.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_upper;
    h_upper.resize(dataBase_.num_surfaces);
    checkCudaErrors(hipMemcpy(h_upper.data(), d_upper, dataBase_.surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_upper\n");
    checkVectorEqual(dataBase_.num_surfaces, upper, h_upper.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_diag;
    h_diag.resize(dataBase_.num_cells);
    checkCudaErrors(hipMemcpy(h_diag.data(), d_diag, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_diag\n");
    checkVectorEqual(dataBase_.num_cells, diag, h_diag.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_source;
    h_source.resize(dataBase_.num_cells);
    checkCudaErrors(hipMemcpy(h_source.data(), d_source, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_source\n");
    checkVectorEqual(dataBase_.num_cells, source, h_source.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_internal_coeffs;
    h_internal_coeffs.resize(dataBase_.num_boundary_surfaces);
    checkCudaErrors(hipMemcpy(h_internal_coeffs.data(), d_internal_coeffs, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_internal_coeffs\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, internal_coeffs, h_internal_coeffs.data(), 1e-14, printFlag);
    DEBUG_TRACE;

    std::vector<double> h_boundary_coeffs;
    h_boundary_coeffs.resize(dataBase_.num_boundary_surfaces);
    checkCudaErrors(hipMemcpy(h_boundary_coeffs.data(), d_boundary_coeffs, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));
    fprintf(stderr, "check h_boundary_coeffs\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_coeffs, h_boundary_coeffs.data(), 1e-14, printFlag);
    DEBUG_TRACE;
}

void dfEEqn::compareHe(const double *he, const double *boundary_he, bool printFlag)
{
    double *h_he = new double[dataBase_.num_cells];
    double *h_boundary_he = new double[dataBase_.num_boundary_surfaces];

    checkCudaErrors(hipMemcpy(h_he, dataBase_.d_he, dataBase_.cell_value_bytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_boundary_he, dataBase_.d_boundary_he, dataBase_.boundary_surface_value_bytes, hipMemcpyDeviceToHost));

    fprintf(stderr, "check h_he\n");
    checkVectorEqual(dataBase_.num_cells, he, h_he, 1e-14, printFlag);
    fprintf(stderr, "check h_boundary_he\n");
    checkVectorEqual(dataBase_.num_boundary_surfaces, boundary_he, h_boundary_he, 1e-14, printFlag);
}
// #endif

void dfEEqn::sync()
{
    checkCudaErrors(hipStreamSynchronize(dataBase_.stream));
}

void dfEEqn::solve()
{
    dataBase_.solve(num_iteration, AMGXSetting::u_setting, d_A, dataBase_.d_he, d_source);
    num_iteration++;
}

void dfEEqn::postProcess(double *h_he, double *h_boundary_he) {}
